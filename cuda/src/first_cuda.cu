#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define DATA_SIZE 1048576
#define THREAD_NUM 256
#define BLOCK_NUM 32

int data[DATA_SIZE];

void GenerateNumbers(int *number, int size){
    for (int i = 0; i < size; i++){
        number[i] = rand() % 10;
    }
}

bool InitCUDA(){
    int count;

    hipGetDeviceCount(&count);
    if (count == 0){
        fprintf(stderr, "There is no device.\n");
        return false;
    }

    int i;
    for (i = 0; i < count; i++){
        hipDeviceProp_t prop;
        if (hipGetDeviceProperties(&prop, i) == hipSuccess){
            if (prop.major >= 1){
                break;
            }
        }
    }

    if (i == count){
        fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
        return false;
    }

    hipSetDevice(i);

    return true;
}

__global__ static void sumOfSquares(int *num, int *result, clock_t *time){

    extern __shared__ int shared[];

    const int tid = threadIdx.x;
    const int bid = blockIdx.x;

    int i;
    int offset = 1, mask = 1;
    if (tid == 0)
        time[bid] = clock();

    shared[tid] = 0;
    for (i = bid * THREAD_NUM + tid; i < DATA_SIZE; i += BLOCK_NUM * THREAD_NUM){
        shared[tid] += num[i] * num[i];
    }

    __syncthreads();
    while(offset < THREAD_NUM){
        if ((tid & mask) == 0){
            shared[tid] += shared[tid + offset];
        }
        offset += offset;
        mask = offset + mask;
        __syncthreads();
    }

    if (tid == 0){
        result[bid] = shared[0];
        time[bid + BLOCK_NUM] = clock();
    }


}

int main(){
    if (!InitCUDA()){
        return 0;
    }

    printf("CUDA initialized.\n");

    GenerateNumbers(data, DATA_SIZE);

    int *gpudata, *result;
    clock_t *time;

    hipMalloc((void**) &gpudata, sizeof(int) * DATA_SIZE);
    hipMalloc((void**) &result, sizeof(int) * THREAD_NUM * BLOCK_NUM);
    hipMalloc((void**) &time, sizeof(clock_t) * BLOCK_NUM * 2);
    hipMemcpy(gpudata, data, sizeof(int) * DATA_SIZE, hipMemcpyHostToDevice);

    sumOfSquares<<<BLOCK_NUM, THREAD_NUM, 0>>>(gpudata, result, time);

    int sum[BLOCK_NUM];
    clock_t time_used[BLOCK_NUM * 2];
    hipMemcpy(&sum, result, sizeof(int) * BLOCK_NUM, hipMemcpyDeviceToHost);
    hipMemcpy(&time_used, time, sizeof(clock_t) * BLOCK_NUM * 2, hipMemcpyDeviceToHost);
    hipFree(gpudata);
    hipFree(result);
    hipFree(time);

    int final_sum = 0;
    for(int i = 0; i < BLOCK_NUM; i++){
        final_sum += sum[i];
    }

    clock_t min_start, max_end;
    min_start = time_used[0];
    max_end = time_used[BLOCK_NUM];
    for (int i = 1; i < BLOCK_NUM; i++){
        if (min_start > time_used[i])
            min_start = time_used[i];
        if (max_end < time_used[i + BLOCK_NUM])
            max_end = time_used[i + BLOCK_NUM];
    }

    printf("sum: %d time: %d\n", final_sum, max_end - min_start);

    final_sum = 0;
    for(int i = 0; i < BLOCK_NUM; i++){
        final_sum += sum[i];
    }
    printf("sum (CPU): %d\n", final_sum);

    return 0;
}
